#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <assert.h>
#include <chrono>

// Function prototypes
void transposeMatrixCPU(float *input, float *output, int rows, int cols);
__global__ void transposeMatrixGPU_Basic(float *input, float *output, int rows, int cols);
__global__ void transposeMatrixGPU_Coalesced(float *input, float *output, int rows, int cols);
__global__ void transposeMatrixGPU_SharedMem(float *input, float *output, int rows, int cols);
void initializeMatrix(float *matrix, int rows, int cols);
bool verifyTranspose(float *original, float *transposed, int rows, int cols);
void printMatrix(float *matrix, int rows, int cols, const char* name);
double measureTimeHR(std::chrono::high_resolution_clock::time_point start, std::chrono::high_resolution_clock::time_point end);
void runPerformanceTest(const char* testName, int rows, int cols);

// Matrix dimensions - starting with 2048x1024, will test multiple sizes
#define DEFAULT_ROWS 2048
#define DEFAULT_COLS 1024
#define TILE_DIM 32
#define BLOCK_ROWS 8

int main()
{
    printf("CUDA Matrix Transposition - Optimized Implementation (High Resolution Timing)\n");
    printf("=========================================================================\n\n");

    // Test different matrix sizes
    printf("Testing multiple matrix sizes:\n");
    printf("==============================\n");
    
    runPerformanceTest("Small Matrix", 512, 256);
    runPerformanceTest("Medium Matrix", 1024, 512);
    runPerformanceTest("Large Matrix", 2048, 1024);
    runPerformanceTest("XLarge Matrix", 4096, 2048);

    return 0;
}

void runPerformanceTest(const char* testName, int rows, int cols)
{
    printf("\n%s (%d x %d):\n", testName, rows, cols);
    printf("----------------------------------------\n");
    
    int matrix_size = rows * cols;
    size_t size = matrix_size * sizeof(float);

    // Use unified memory for easier management and potentially better performance
    float *unified_input, *unified_output_basic, *unified_output_coalesced, *unified_output_shared;
    float *h_output_cpu;

    // Allocate unified memory
    hipMallocManaged(&unified_input, size);
    hipMallocManaged(&unified_output_basic, size);
    hipMallocManaged(&unified_output_coalesced, size);
    hipMallocManaged(&unified_output_shared, size);
    
    // Allocate host memory for CPU implementation
    h_output_cpu = (float*)malloc(size);

    if (!h_output_cpu) {
        fprintf(stderr, "Memory allocation failed!\n");
        return;
    }

    // Initialize input matrix
    initializeMatrix(unified_input, rows, cols);

    // CPU Implementation
    auto cpu_start = std::chrono::high_resolution_clock::now();
    transposeMatrixCPU(unified_input, h_output_cpu, rows, cols);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    double cpu_time = measureTimeHR(cpu_start, cpu_end);

    // GPU Implementation 1: Basic (from Part 1)
    dim3 blockSize_basic(16, 16);
    dim3 gridSize_basic((cols + blockSize_basic.x - 1) / blockSize_basic.x, 
                       (rows + blockSize_basic.y - 1) / blockSize_basic.y);

    auto gpu_basic_start = std::chrono::high_resolution_clock::now();
    transposeMatrixGPU_Basic<<<gridSize_basic, blockSize_basic>>>(unified_input, unified_output_basic, rows, cols);
    hipDeviceSynchronize();
    auto gpu_basic_end = std::chrono::high_resolution_clock::now();
    double gpu_basic_time = measureTimeHR(gpu_basic_start, gpu_basic_end);

    // GPU Implementation 2: Coalesced Memory Access  
    dim3 blockSize_coalesced(TILE_DIM, BLOCK_ROWS);
    dim3 gridSize_coalesced((cols + TILE_DIM - 1) / TILE_DIM, 
                           (rows + TILE_DIM - 1) / TILE_DIM);

    auto gpu_coalesced_start = std::chrono::high_resolution_clock::now();
    transposeMatrixGPU_Coalesced<<<gridSize_coalesced, blockSize_coalesced>>>(unified_input, unified_output_coalesced, rows, cols);
    hipDeviceSynchronize();
    auto gpu_coalesced_end = std::chrono::high_resolution_clock::now();
    double gpu_coalesced_time = measureTimeHR(gpu_coalesced_start, gpu_coalesced_end);

    // GPU Implementation 3: Shared Memory Optimization
    auto gpu_shared_start = std::chrono::high_resolution_clock::now();
    transposeMatrixGPU_SharedMem<<<gridSize_coalesced, blockSize_coalesced>>>(unified_input, unified_output_shared, rows, cols);
    hipDeviceSynchronize();
    auto gpu_shared_end = std::chrono::high_resolution_clock::now();
    double gpu_shared_time = measureTimeHR(gpu_shared_start, gpu_shared_end);

    // Verify correctness
    bool cpu_correct = verifyTranspose(unified_input, h_output_cpu, rows, cols);
    bool gpu_basic_correct = verifyTranspose(unified_input, unified_output_basic, rows, cols);
    bool gpu_coalesced_correct = verifyTranspose(unified_input, unified_output_coalesced, rows, cols);
    bool gpu_shared_correct = verifyTranspose(unified_input, unified_output_shared, rows, cols);

    printf("Correctness: CPU=%s, GPU_Basic=%s, GPU_Coalesced=%s, GPU_Shared=%s\n",
           cpu_correct ? "✓" : "✗",
           gpu_basic_correct ? "✓" : "✗",
           gpu_coalesced_correct ? "✓" : "✗",
           gpu_shared_correct ? "✓" : "✗");

    // Performance Results
    printf("\nPerformance Results:\n");
    printf("CPU Time:           %.3f ms (%.2f GB/s)\n", cpu_time, (2.0 * size) / (cpu_time * 1e-3) / 1e9);
    printf("GPU Basic:          %.3f ms (%.2f GB/s) - Speedup: %.2fx\n", gpu_basic_time, (2.0 * size) / (gpu_basic_time * 1e-3) / 1e9, cpu_time / gpu_basic_time);
    printf("GPU Coalesced:      %.3f ms (%.2f GB/s) - Speedup: %.2fx\n", gpu_coalesced_time, (2.0 * size) / (gpu_coalesced_time * 1e-3) / 1e9, cpu_time / gpu_coalesced_time);
    printf("GPU Shared Mem:     %.3f ms (%.2f GB/s) - Speedup: %.2fx\n", gpu_shared_time, (2.0 * size) / (gpu_shared_time * 1e-3) / 1e9, cpu_time / gpu_shared_time);

    // Cleanup
    free(h_output_cpu);
    hipFree(unified_input);
    hipFree(unified_output_basic);
    hipFree(unified_output_coalesced);
    hipFree(unified_output_shared);
}

// CPU matrix transpose implementation
void transposeMatrixCPU(float *input, float *output, int rows, int cols)
{
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            output[j * rows + i] = input[i * cols + j];
        }
    }
}

// Basic GPU matrix transpose kernel (from Part 1)
__global__ void transposeMatrixGPU_Basic(float *input, float *output, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        int input_idx = row * cols + col;
        int output_idx = col * rows + row;
        output[output_idx] = input[input_idx];
    }
}

// Fixed coalesced memory access version
__global__ void transposeMatrixGPU_Coalesced(float *input, float *output, int rows, int cols)
{
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    
    int width = cols;
    int height = rows;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        if (xIndex < width && (yIndex + j) < height) {
            int input_idx = (yIndex + j) * width + xIndex;
            int output_idx = xIndex * height + (yIndex + j);
            output[output_idx] = input[input_idx];
        }
    }
}

// Shared memory optimized version
__global__ void transposeMatrixGPU_SharedMem(float *input, float *output, int rows, int cols)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1]; // +1 to avoid bank conflicts

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = cols;
    int height = rows;

    // Load data into shared memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        if (x < width && (y + j) < height) {
            tile[threadIdx.y + j][threadIdx.x] = input[(y + j) * width + x];
        }
    }

    __syncthreads();

    // Calculate transposed coordinates
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    // Write transposed data from shared memory to global memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        if (x < height && (y + j) < width) {
            output[(y + j) * height + x] = tile[threadIdx.x][threadIdx.y + j];
        }
    }
}

// Initialize matrix with random values
void initializeMatrix(float *matrix, int rows, int cols)
{
    srand(time(NULL));
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = (float)rand() / RAND_MAX * 100.0f;
    }
}

// Verify transpose correctness
bool verifyTranspose(float *original, float *transposed, int rows, int cols)
{
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            float orig = original[i * cols + j];
            float trans = transposed[j * rows + i];
            if (abs(orig - trans) > 1e-5) {
                printf("Mismatch at (%d,%d): orig=%.6f, trans=%.6f\n", i, j, orig, trans);
                return false;
            }
        }
    }
    return true;
}

// Print matrix (for debugging small matrices)
void printMatrix(float *matrix, int rows, int cols, const char* name)
{
    if (rows > 10 || cols > 10) {
        printf("%s: Matrix too large to print (size: %dx%d)\n", name, rows, cols);
        return;
    }
    
    printf("%s:\n", name);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.2f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Measure time in milliseconds using high resolution clock
double measureTimeHR(std::chrono::high_resolution_clock::time_point start, std::chrono::high_resolution_clock::time_point end)
{
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    return duration.count() / 1000.0; // Convert to milliseconds
}
